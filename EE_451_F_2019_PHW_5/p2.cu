
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define DIM_GRID 32
#define DIM_BLOCK 32
#define SIZE 1024

__global__ void matrix_multify(int *a, int *b, int*c) {
    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
    if ((row < SIZE)&&(col < SIZE)) {
        int value_C = 0;
        for (int tileId=0; tileId<(SIZE/DIM_GRID); tileId++) {
            __shared__ int a_tile[SIZE/DIM_GRID][SIZE/DIM_GRID];
            __shared__ int b_tile[SIZE/DIM_GRID][SIZE/DIM_GRID];
            //loading the a and b tiles into shared memory
            a_tile[threadIdx.y][threadIdx.x] = a[row*SIZE+(tileId*SIZE/DIM_GRID+threadIdx.x)];
            b_tile[threadIdx.y][threadIdx.x] = b[(tileId*SIZE/DIM_GRID+threadIdx.y)*SIZE+col];
            __syncthreads();
            for (int k=0; k<SIZE/DIM_GRID; k++) {
                value_C+=a_tile[threadIdx.y][k]*b_tile[k][threadIdx.x];
            }
            __syncthreads();
        }
        c[row*SIZE+col] = value_C;
    }
}
int main() {
    int i;
    struct timespec start, stop;
    double time;
    int *a = (int*)malloc(sizeof(int)*SIZE*SIZE);
    int *b = (int*)malloc(sizeof(int)*SIZE*SIZE);
    int *c = (int*)malloc(sizeof(int)*SIZE*SIZE);

    for (i=0; i<SIZE*SIZE; i++) {
        a[i] = 1;
        b[i] = 2;
        c[i] = 0;
    }

    int *a_gpu, *b_gpu, *c_gpu;
    hipMalloc((void**)&a_gpu, sizeof(int)*SIZE*SIZE);
    hipMalloc((void**)&b_gpu, sizeof(int)*SIZE*SIZE);
    hipMalloc((void**)&c_gpu, sizeof(int)*SIZE*SIZE);

    hipMemcpy(a_gpu, a, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);

    dim3 dimBlock(DIM_BLOCK, DIM_BLOCK);
    dim3 dimGrid(DIM_GRID, DIM_GRID);
    //start our cuda program
    if (clock_gettime(CLOCK_REALTIME, &start) == -1) {
        perror("clock gettime");
    }
    matrix_multify<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, c_gpu);
    hipMemcpy(c, c_gpu, sizeof(int)*SIZE*SIZE, hipMemcpyDeviceToHost);

    if(clock_gettime(CLOCK_REALTIME, &stop) == -1) {
        perror("clock gettime");
    }
    time = (stop.tv_sec - start.tv_sec)+(double)(stop.tv_nsec-start.tv_nsec)/1e9;
    printf("time is %f ns\n", time*1e9);
    printf("c[451][451] is %d\n", c[451*SIZE+451]);

    free(a);
    free(b);
    free(c);
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
    return 0;
}