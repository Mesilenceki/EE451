#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define SIZE 1024
#define GRID_SIZE 64;
#define BLOCK_SIZE 16;

__global__ void matrix_mupltify(int *a, int *b, int *c) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int c_val = 0;
    for (int k=0; k<SIZE; k++) {
        c[row*SIZE+col] += a[row*SIZE+k]*b[k*SIZE+col];
    }
    c[row*SIZE+col] = c_val;
}

int main() {
    int i=0;
    
    int *a = (int*)malloc(sizeof(int)*SIZE*SIZE);
    int *b = (int*)malloc(sizeof(int)*SIZE*SIZE);
    int *c = (int*)malloc(sizeof(int)*SIZE*SIZE);

    for (i=0; i<SIZE*SIZE; i++) {
        a[i] = 1;
        b[i] = 2;
    }

    int *gpu_a, *gpu_b, *gpu_c;
    hipMalloc((void**)&gpu_a, sizeof(int)*SIZE*SIZE);
    hipMalloc((void**)&gpu_b, sizeof(int)*SIZE*SIZE);
    hipMalloc((void**)&gpu_c, sizeof(int)*SIZE*SIZE);

    hipMemcpy(gpu_a, a, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    //start our cuda program
    if (clock_gettime(CLOCK_REALTIME, &start) == -1) {
        perror("clock gettime");
    }
    
    matrix_mupltify<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
    hipMemcpy(c, gpu_c, sizeof(int)*SIZE*SIZE, hipMemcpyDeviceToHost);
    
    if(clock_gettime(CLOCK_REALTIME, &stop) == -1) {
        perror("clock gettime");
    }
    time = (stop.tv_sec - start.tv_sec)+(double)(stop.tv_nsec-start.tv_nsec)/1e9;
    printf("time is %f ns\n", time*1e9);
    printf("c[451][451]=%d\n", c[451*SIZE+451]);

    free(a);
    free(b);
    free(c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    return 0;
}