#include "hip/hip_runtime.h"

#define SIZE 1000
#define DIM_GRID 10
#define DIM_BLOCK 100
//kernel function
__global__ void multify(int *a, in*b, int *c) {
    //a is the 1K x 1K matrix, b is 1K vector, c is 1K vector too
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    float c_val = 0;
    for (int tileId=0; tileId<(DIM_GRID; tileId++)) {
        __shared__ tile_b[SIZE/DIM_GRID];
        tile_b = b[(tileId*threadIdx.x)*SIZE+col];
        __syncthreads();
        for (int k=0; k<SIZE; k++) {
            c_val+=a[threadIdx.y][k]*tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    c[row*SIZE+col] = c_val;
}


//CUDA code on the host side
int main() {
    //initialize a, b, c
    int *a = (int*)malloc(sizeof(int)*SIZE*SIZE);
    int *b = (int*)malloc(sizeof(int)*SIZE);
    int *c = (int*)malloc(sizeof(int)*SIZE);

    int *gpu_a, *gpu_b, *gpu_c;
    cudamalloc((void**)&gpu_a, sizeof(int)*SIZE*SIZE);
    cudamalloc((void**)&gpu_b, sizeof(int)*SIZE);
    cudamalloc((void**)&gpu_c, sizeof(int)*SIZE);

    hipMemcpy(gpu_a, a, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(int)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(gpu_c, c, sizeof(int)*SIZE, hipMemcpyHostToDevice);

    dim3 dimGrid(DIM_GRID);
    dim3 dimBlock(DIM_BLOCK);
    multify<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
    hipMemcpy(c, gpu_c, sizeof(int)*SIZE, hipMemcpyDeviceToHost); 
}